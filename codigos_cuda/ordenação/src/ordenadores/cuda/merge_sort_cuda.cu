// ============================================================
//                  Observações gerais
// ============================================================
/* 
    Este arquivo implementa um merge sort paralelo executado na GPU usando CUDA. O fluxo geral é:
        1. Ler vetores de arquivos binários (int)
        2. Copiar os dados para a GPU
        3. Executar várias etapas de "merge" (tamanho da sublista dobrando a cada iteração)
        4. Copiar resultado de volta para o host e regravar o arquivo
        5. Registrar tempos em CSV
*/ 


#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define THREADS_POR_BLOCO 256

/*
    THREADS_POR_BLOCO: número de threads por bloco CUDA quando lançamos um kernel.
        - É uma configuração de desempenho que normalmente depende da GPU.
        - não necessariamente mais threads significa mais desempenho.
*/ 

using namespace std;

// ============================================================
//                  KERNEL DO MERGE SORT (GPU)
// ============================================================

/*
    MergeKernel: kernel CUDA que faz o "merge" entre dois blocos
    já ordenados do array.

    Parâmetros:
        - dados: ponteiro para o vetor principal (na memória do device)
        - buffer_temp:  ponteiro para área temporária (na memória do device)
        - N: tamanho total do array (número de elementos)
        - tamanho_atual: tamanho atual da sublista ordenada. A cada iteração do
            MergeSortCuda esta variável dobra (1,2,4,8,...)
    Funcionamento:
        - Cada thread calcula qual par de sublistas deve mesclar baseado no seu ID.
        - As sublistas a serem mescladas são [esquerda, meio) e [meio, direita)
        - Utiliza um merge clássico para combinar as duas sublistas em buffer_temp.
        - Finalmente, copia o resultado de volta para o array original `dados`.

    Cada thread é responsável por mesclar duas sublistas adjacentes
    de tamanho tamanho_atual (ou menores se estivermos nas bordas).
 */
__global__ void MergeKernel(int* dados, int* buffer_temp, int N, int tamanho_atual)
{
    // thread_id: índice global da thread (0..num_threads-1)
    long thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Calcula os índices (em elementos) das duas metades a serem mescladas:
    long esquerda  = (long)thread_id * tamanho_atual * 2L;      // esquerda  = início da primeira metade
    long meio   = esquerda + tamanho_atual;                     // meio   = início da segunda metade
    long direita = (long)(thread_id + 1) * tamanho_atual * 2L;  // direita = fim (exclusivo) da segunda metade

    // Verificações de segurança para evitar acessar além do array.
    if (esquerda >= N) return;        // nada a fazer se o início estiver fora
    if (meio > N)  meio = N;          // ajusta meio se ultrapassar
    if (direita > N) direita = N;     // ajusta direita se ultrapassar
    if (esquerda >= direita) return;  // segmento inválido (tamanho 0)


    long idx_esq = esquerda;     // idx_esq: ponteiro corrente na metade esquerda
    long idx_dir = meio;         // idx_dir: ponteiro corrente na metade direita
    long idx_atual = esquerda;   // idx_atual: posição corrente onde iremos colocar no buffer_temp


    /* 
        Merge clássico: 
        - Compara o elemento mais à frente das duas metades e escreve o menor em buffer_temp. 
        - Continua até que uma das metades acabe ou até preencher todo o intervalo [esquerda,direita).
    */ 
    while (idx_esq < meio && idx_dir < direita && idx_atual < direita) 
    {
        int a = dados[idx_esq];
        int b = dados[idx_dir];
        if (a <= b) 
        {
            buffer_temp[idx_atual++] = a;
            idx_esq++;
        } else {
            buffer_temp[idx_atual++] = b;
            idx_dir++;
        }
    }

    // Se sobrou elementos na metade esquerda, copia para buffer_temp
    while (idx_esq < meio && idx_atual < direita)
    {
        buffer_temp[idx_atual++] = dados[idx_esq++];
    }

    // Se sobrou elementos na metade direita, copia para buffer_temp
    while (idx_dir < direita && idx_atual < direita)
    {
        buffer_temp[idx_atual++] = dados[idx_dir++];
    }
        

    // Copia o intervalo mesclado de volta para o array original `dados`.
    // Fazemos isso aqui no final do kernel para manter `dados` sempre consistente
    // para iterações futuras (essa escolha evita manter alternância entre dois
    // buffers na CPU; no entanto, dependências de escrita/leitura precisam ser
    // consideradas — aqui cada thread escreve apenas no seu segmento).
    for (long i = esquerda; i < direita && i < N; ++i)
    {
        dados[i] = buffer_temp[i];
    }
       
}

// ============================================================
//          FUNÇÃO DE CONTROLE DO MERGE SORT (GPU)
// ============================================================

/*
    MergeSortCuda: coordena as chamadas ao kernel para ordenar todo o vetor.
 
    Parâmetros:
        - dados: ponteiro para o vetor no device (GPU)
        - buffer_temp:  ponteiro para buffer temporário no device
        - N: tamanho do vetor
 
    Funcionamento:
        - A ideia do merge sort bottom-up: começamos com sublistas de tamanho 1,
        depois 2, 4, 8,... até cobrir todo o vetor. Em cada passo, lançamos
        um kernel onde cada thread junta (merge) duas sublistas adjacentes de
        tamanho `tamanho`.
 */
void MergeSortCuda(int* dados, int* buffer_temp, int N)
{
    if (N <= 1) 
    {
        return;
    }

    // 'tamanho' é o tamanho atual das sublistas ordenadas (1,2,4,8...)
    for (int tamanho = 1; tamanho < N; tamanho <<= 1) 
    {
        // Calcula quantas threads precisamos: cada thread faz o merge de
        // duas sublistas de tamanho `tamanho` => cobre `tamanho*2` elementos.
        long total_threads = (N + (tamanho * 2L - 1L)) / (tamanho * 2L);

        // Converte número de threads em número de blocos considerando
        // THREADS_POR_BLOCO por bloco.
        int blocos = (int)((total_threads + THREADS_POR_BLOCO - 1) / THREADS_POR_BLOCO);

        // Lança o kernel com a configuração (blocos, THREADS_POR_BLOCO).
        MergeKernel<<<blocos, THREADS_POR_BLOCO>>>(dados, buffer_temp, N, tamanho);

        // Sincroniza e verifica erros de execução. cudaDeviceSynchronize
        // bloqueia até que o kernel termine — importante para checar erros
        // e para garantir que a próxima iteração trabalhe com dados consistentes.
        hipError_t syncErr = hipDeviceSynchronize();
        if (syncErr != hipSuccess) 
        {
            fprintf(stderr, "Erro após cudaDeviceSynchronize() na iteração tamanho=%d: %s\n", tamanho, hipGetErrorString(syncErr));
            hipError_t launchErr = hipGetLastError();
            if (launchErr != hipSuccess)
            {
                fprintf(stderr, "Erro de launch: %s\n", hipGetErrorString(launchErr));
            }

            return; // Em caso de erro, aborta a ordenação
        }

        // Checa erros residuais do lançamento do kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) 
        {
            fprintf(stderr, "Erro no kernel após tamanho=%d: %s\n", tamanho, hipGetErrorString(err));
            return;
        }
    }
}

// ============================================================
//    FUNÇÃO HostParaDevice() - GERENCIA COPIAS HOST/DEVICE
// ============================================================

/*
    HostParaDevice: interface que recebe os dados no host (CPU), aloca memória na GPU,
    copia os dados para lá, executa MergeSortCuda na GPU e copia o resultado
    de volta para o host.
 
    Parâmetros:
        - dados_host: ponteiro para array de inteiros na memória do host
        - N: número de elementos no array
 */
void HostParaDevice(int* dados_host, int N)
{
    int *dados_device = nullptr;
    int *buffer_device = nullptr; // ponteiros para device (GPU)

    hipError_t err;

    // Aloca memória para `dados_device` na GPU: espaço para N inteiros
    err = hipMalloc(&dados_device, N * sizeof(int));
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Erro ao alocar dados_device (N=%d): %s\n", N, hipGetErrorString(err));
        return; // aborta se não conseguiu alocar
    }

    // Aloca memória para buffer temporário `buffer_device` na GPU
    err = hipMalloc(&buffer_device, N * sizeof(int));
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Erro ao alocar buffer_device (N=%d): %s\n", N, hipGetErrorString(err));
        hipFree(dados_device);
        return;
    }

    // Copia dados do host (CPU) para o device (GPU)
    err = hipMemcpy(dados_device, dados_host, N * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Erro em hipMemcpy Host->Device (N=%d): %s\n", N, hipGetErrorString(err));
        hipFree(dados_device);
        hipFree(buffer_device);
        return;
    }

    // Chama a rotina que executa o merge sort na GPU (controlada por MergeSortCuda)
    MergeSortCuda(dados_device, buffer_device, N);

    // Copia o resultado ordenado de volta para o host
    err = hipMemcpy(dados_host, dados_device, N * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Erro em hipMemcpy Device->Host (N=%d): %s\n", N, hipGetErrorString(err));
    }

    // Libera memória alocada no device
    hipFree(dados_device);
    hipFree(buffer_device);
}

// ============================================================
//             FUNÇÕES AUXILIARES DE ENTRADA/SAÍDA
// ============================================================

/*
    ExecMergeCuda: dado um conjunto de caminhos para arquivos binários contendo
    inteiros, realiza a ordenação utilizando a GPU e grava tempos em CSV.

    Parâmetros:
        - entradas: array de caminhos (const char*) para arquivos binários
        - num_entradas: número de entradas no array
        - csv_saida: caminho do arquivo CSV de saída onde serão registrados os tempos

    Funcionamento:
        - Para cada arquivo de entrada:
        - Abre o arquivo em modo binário
        - Determina o número de inteiros no arquivo
        - Lê os inteiros para um vetor alocado dinamicamente
        - Mede o tempo de ordenação chamando HostParaDevice (que usa a GPU)
        - Regrava o arquivo com os dados ordenados
        - Registra o tempo no arquivo CSV
        - Usa fseek/ftell para descobrir o tamanho do arquivo (em bytes) e divide
          por sizeof(int) para obter o número de inteiros.
        - Lê todo o arquivo para um vetor alocado dinamicamente (new int[tamanho]).
 */
void ExecMergeCuda(const char **entradas, int num_entradas, const char *csv_saida)
{
    FILE *csv = fopen(csv_saida, "a");
    if (!csv) 
    {
        perror("Erro ao abrir arquivo CSV");
        return;
    }

    for (int i = 0; i < num_entradas; i++) 
    {
        FILE *file = fopen(entradas[i], "rb+"); // abre para leitura/escrita binária
        if (!file) 
        {
            perror(entradas[i]);
            return;
        }

        // Determina quantos inteiros existem no arquivo
        fseek(file, 0, SEEK_END);
        long tamanho = ftell(file) / sizeof(int);
        fseek(file, 0, SEEK_SET);

        int *v = new int[tamanho]; // aloca vetor no heap para os dados
        if (fread(v, sizeof(int), tamanho, file) != (size_t)tamanho) 
        {
            perror("Erro ao ler o arquivo");
            fclose(file);
            delete[] v;
            continue;
        }

        // Mede o tempo de ordenação usando chrono (CPU side timer)
        auto start = chrono::high_resolution_clock::now();
        HostParaDevice(v, tamanho); // ordena chamando a GPU
        auto end = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed = end - start;
        double tempo = elapsed.count();

        printf("MergeSort CUDA - Tempo para ordenar %s: %f s\n", entradas[i], tempo);
        fprintf(csv, "MergeSort - CUDA,%ld,%f\n", tamanho, tempo);

        // Regrava o arquivo com os valores ordenados (volta ao início com fseek)
        fseek(file, 0, SEEK_SET);
        if(fwrite(v, sizeof(int), tamanho, file) != (size_t)tamanho)
        {
            perror("Erro ao escrever no arquivo");
            fclose(file);
            delete[] v;
            continue;
        }

        fclose(file);
        delete[] v; // libera o vetor do host
    }

    fclose(csv);
}

